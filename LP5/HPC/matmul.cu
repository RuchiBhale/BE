#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;
using namespace std::chrono;
#define block_size 16
void initialize(int *arr, int rows,int cols){
  for(int i=0;i<rows;i++){
    for(int j=0;j<cols;j++){
      arr[i*cols+j] = rand()%100+1;
    }
  }
}
void print(int *arr,int rows,int cols){
  for(int i=0;i<rows;i++){
    for(int j=0;j<cols;j++){
      cout<<arr[i*cols+j]<<" ";
    }
  }
}
void matmul(int *a, int *b,int *c, int common,int c_rows,int c_cols){
  for(int i=0;i<c_rows;i++){
    for(int j=0;j<c_cols;j++){
      int sum=0;
      for(int k=0;k<common;k++){
        sum+= a[i*common+k]*b[k*c_cols+j];

      }
      c[i*c_cols+j] = sum;

    }
  }
}
__global__ void gpu_mul(int *a, int *b,int *c, int common, int c_rows, int c_cols){
  int rows = blockIdx.y * blockDim.y + threadIdx.y;
  int cols = blockIdx.x * blockDim.x +threadIdx.x;
  if(rows<c_rows && cols < c_cols){
    int sum =0;
    for(int j=0;j<common;j++){
      sum+=a[rows*common+j]*b[j*c_cols+cols];
    }
    c[rows*c_cols+cols] = sum;
  }
}
int main(){
  int *cpu_a,*cpu_b,*cpu_c;
  int a_rows, a_cols, b_rows,b_cols;
  cout<<"\nEnter the no of rows in A: ";
  cin>>a_rows;
  cout<<"\nEnter the number of cols in A: ";
  cin>>a_cols;
  cout<<"\nEnter B rows: ";
  cin>> b_rows;
  cout<<"\nEnter B cols: ";
  cin>>b_cols;
  int a_size = a_rows*a_cols;
  int b_size = b_rows *b_cols;
  int c_size = a_rows * b_cols;
  cpu_a = new int[a_size];
  cpu_b = new int[b_size];
  cpu_c = new int[c_size];
  initialize(cpu_a, a_rows,a_cols);
  initialize(cpu_b, b_rows,b_cols);
  int *gpu_a, *gpu_b, *gpu_c;
  int vectorbytes_a = a_size *sizeof(int);
  int vectorbytes_b = b_size *sizeof(int);
  int vectorbytes_c = c_size * sizeof(int);

  hipMalloc(&gpu_a, vectorbytes_a);
  hipMalloc(&gpu_b, vectorbytes_b);
  hipMalloc(&gpu_c, vectorbytes_c);

  cout<<"\nMatrix A: ";
  print(cpu_a,a_rows, a_cols);
  cout<<"\nMatrix B: ";
  print(cpu_b, b_rows, b_cols);

  hipMemcpy(gpu_a, cpu_a,vectorbytes_a, hipMemcpyHostToDevice);
  hipMemcpy(gpu_b, cpu_b, vectorbytes_b, hipMemcpyHostToDevice);
  dim3 dimblock(block_size*block_size);
  dim3 gridDim ((a_rows+block_size-1)/ block_size, (b_cols+block_size-1)/block_size);

  auto start= high_resolution_clock::now();
  gpu_mul<<<gridDim, dimblock>>>(gpu_a,gpu_b,gpu_c,a_cols,a_rows,b_cols);
  auto end = high_resolution_clock::now();
  auto duration = duration_cast<microseconds> (end-start);
  hipMemcpy(cpu_c,gpu_c, vectorbytes_c, hipMemcpyDeviceToHost);
  cout<<"gpu Multiplication is: "<<endl;
  print(cpu_c, a_rows,b_cols);
  cout<<"\nGPU time: "<<duration.count()<<" microseconds\n"<<endl;

  hipFree(gpu_a);hipFree(gpu_b);hipFree(gpu_c);
  start = high_resolution_clock::now();
  matmul(cpu_a,cpu_b,cpu_c,a_cols,a_rows,b_cols);
  end = high_resolution_clock::now();
  duration = duration_cast<microseconds>(end-start);
  cout<<"CPU Multiplication: "<<endl;
  print(cpu_c, a_rows, b_cols);
  cout<<"\nCPU time: "<<duration.count()<<" microseconds\n";

  free(cpu_a);free(cpu_b);free(cpu_c);
  return 0;


}
