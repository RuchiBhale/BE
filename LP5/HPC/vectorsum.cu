#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;
using namespace std::chrono;
#define block_size 16
void initialize(int *arr, int n){
  for(int i=0;i<n;i++){
    arr[i] = rand()%100+1;
  }
}
void print(int *arr, int n){
  for(int i=0;i<n;i++){
    cout<<arr[i]<<" ";
  }
  cout<<endl;
}
void cpu_sum(int *a, int *b, int *c,int n){
  for(int i=0;i<n;i++){
    c[i] = a[i] + b[i];
  }
}
__global__ void gpu_sum(int *a,int*b,int*c,int n){
  int id = blockIdx.x *blockDim.x + threadIdx.x;
  if(id<n){
    c[id] = a[id] + b[id];
  }
}
int main(){
  int n;
  cout<<"Enter the size: ";
  cin>>n;
  int *cpu_a,*cpu_b,*cpu_c;
  int *gpu_a,*gpu_b,*gpu_c;
  cpu_a = new int[n];
  cpu_b = new int[n];
  cpu_c = new int[n];
  int vectorbytes = n*sizeof(int);

  initialize(cpu_a, n);
  initialize(cpu_b,n);
  cout<<"Vector A: ";
  print(cpu_a,n);
  cout<<"Vector B: ";
  print(cpu_b,n);
  hipMalloc(&gpu_a, vectorbytes);
  hipMalloc(&gpu_b, vectorbytes);
  hipMalloc(&gpu_c, vectorbytes);
  hipMemcpy(gpu_a, cpu_a, vectorbytes, hipMemcpyHostToDevice);
  hipMemcpy(gpu_b, cpu_b, vectorbytes, hipMemcpyHostToDevice);
  dim3 dimblock(block_size);
  dim3 gridDim((n+block_size-1)/block_size);
  auto start = high_resolution_clock::now();
  gpu_sum<<<gridDim, dimblock>>>(gpu_a,gpu_b,gpu_c,n);
  auto end = high_resolution_clock::now();
  hipMemcpy(cpu_c, gpu_c,vectorbytes,hipMemcpyDeviceToHost);
  cout<<"GPU sum: "<<endl;
  print(cpu_c,n);
  auto duration = duration_cast<microseconds>(end-start);
  cout<<"GPU time: "<<duration.count()<<" microseconds\n";

  hipFree(gpu_a); hipFree(gpu_b); hipFree(gpu_c);
  start = high_resolution_clock::now();
  cpu_sum(cpu_a,cpu_b,cpu_c,n);
  end = high_resolution_clock::now();
  duration = duration_cast<microseconds>(end-start);
  cout<<"CPU sum: "<<endl;
  print(cpu_c,n);
  cout<<"CPU time is: "<<duration.count()<<" microseconds\n";
  free(cpu_a); free(cpu_b); free(cpu_c);
  return 0;
}
